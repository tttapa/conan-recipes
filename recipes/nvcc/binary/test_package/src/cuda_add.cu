#include "hip/hip_runtime.h"
#include "cuda_add.hpp"
#include <hip/hip_runtime.h>

__global__ void vector_add(const int *a, const int *b, int *result, int N) {
    int idx = threadIdx.x + blockIdx.x * blockDim.x;
    if (idx < N) {
        result[idx] = a[idx] + b[idx];
    }
}

void launch_cuda_add(const int *a, const int *b, int *result, int N) {
    int *d_a = nullptr;
    int *d_b = nullptr;
    int *d_result = nullptr;

    hipMalloc(&d_a, N * sizeof(int));
    hipMalloc(&d_b, N * sizeof(int));
    hipMalloc(&d_result, N * sizeof(int));

    hipMemcpy(d_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(d_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    int threadsPerBlock = 256;
    int blocks = (N + threadsPerBlock - 1) / threadsPerBlock;
    vector_add<<<blocks, threadsPerBlock>>>(d_a, d_b, d_result, N);

    hipMemcpy(result, d_result, N * sizeof(int), hipMemcpyDeviceToHost);

    hipFree(d_a);
    hipFree(d_b);
    hipFree(d_result);
}
